#include <stdio.h>
#include <assert.h>
#include <stdlib.h>
#include <iostream>
#include <numeric>
#include <math.h>
#include <hip/hip_runtime.h>



int block = 1024;
int thread = 1024;
__device__ int mandel(float cr, float ci);

__device__ int mandel(float cr, float ci){
	float zr=0, zi=0, zr2=0, zi2=0;
	int i;
	for(i=1;i<256;i++){
		zi=zr*zi;
		zi+=zi;
		zi+=ci;
		//zi=2*zr*zi+ci;
		zr=zr2-zi2+cr;
		zr2=zr*zr;
		zi2=zi*zi;
		if(zr2+zi2>4)
			break;
	}
	//printf("yo %d\n",i);
	return i;
}

__global__ void kernel(int *arr, int width, int height, float Xmin, float Ymin, float Xinc, float Yinc){


	float px_per_thread = width*height/(gridDim.x*blockDim.x);
	float index = blockDim.x*blockIdx.x+threadIdx.x;
	float offset = px_per_thread*index;
	for(int i=offset; i<offset+px_per_thread;i++){
		int x=i%width;
		int y=i/width;
		float cr=Xmin+x*Xinc;
		float ci=Ymin+y*Yinc;
		arr[y*width + x] = mandel(cr, ci);

	}
	//if(gridDim.x * blockDim.x * px_per_thread < width*height && index < (width*height) - (blockDim.x * gridDim.x)){
		//int i = blockDim.x * gridDim.x * px_per_thread + index;
		//int x = i%width;
		//int y = i/width;
		//float cr = xmin + x*0.00293;
		//float ci = ymin + y*0.0039;
		//arr[y*width+x] = mandel(cr, ci);
		//arr[y*width+x] = 500;
	//}
	//printf("index %f\n", index);
}

int main(int argc, char *argv[]){
    clock_t tic = clock();
    if(argc != 4){
        printf("Enter width, height and filename\n");
        return 1;
    }
    int width = atoi(argv[1]);
    int height = atoi(argv[2]);
	int size = width*height*sizeof(int);
	int *ar;
	int *d_arr;
	float Xmin = -2, Xmax = 1, Ymin = -1.5, Ymax = 1.5;
	float Xinc = (Xmax-Xmin)/width;
	float Yinc = (Ymax-Ymin)/height;

    //cuda memory
	hipMalloc((void**) &d_arr, size);

	//host memory
    ar = (int*)malloc(size);

    //run cuda
	kernel<<<block, thread>>>(d_arr, width, height, Xmin, Ymin, Xinc, Yinc);

	hipMemcpy(ar, d_arr, size, hipMemcpyDeviceToHost);

	//Create and write output
	FILE *fp;
	fp = (fopen(argv[3],"w"));
	if(fp==NULL){
		printf("Error!");
		exit(1);
	}
	fprintf(fp,"%d %d\n", width, height);
	for(int i=0; i<(width*height);i++){
			fprintf(fp,"%d ",ar[i]);
	}

	hipFree(d_arr);
	free(ar);
    clock_t toc = clock();

    float time_spent = (float)(toc-tic)/CLOCKS_PER_SEC;
    printf("CUDA Execution Time %f sec\n", time_spent);
}

